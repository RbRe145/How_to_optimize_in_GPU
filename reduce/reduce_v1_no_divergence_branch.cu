#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256

// bank conflict
template<typename T>
__global__ void reduce1(T* input, T* output){
    __shared__ T shmem[THREAD_PER_BLOCK];
    int tid = threadIdx.x;
    int src_i = blockDim.x*blockIdx.x + threadIdx.x;
    shmem[tid] = input[src_i];
    int warp_size = 128 / sizeof(T);
    __syncthreads();
    for(int i=blockDim.x/2; i>=1; i>>=1){
        if(tid < i){
            shmem[tid] += shmem[tid+i];
        }
        // i<=warp_size的时候，只有warp0在工作，不需要同步其他warp内的线程
        if(i>warp_size){
            __syncthreads();
        }
    }
    if(tid==0) output[blockIdx.x] = shmem[tid];
}
bool check(float *out,float *res,int n){
    for(int i=0;i<n;i++){
        if(out[i]!=res[i])
            return false;
    }
    return true;
}

int main(){
    const int N=32*1024*1024;
    float *a=(float *)malloc(N*sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a,N*sizeof(float));

    int block_num=N/THREAD_PER_BLOCK;
    float *out=(float *)malloc((N/THREAD_PER_BLOCK)*sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out,(N/THREAD_PER_BLOCK)*sizeof(float));
    float *res=(float *)malloc((N/THREAD_PER_BLOCK)*sizeof(float));

    for(int i=0;i<N;i++){
        a[i]=1;
    }

    for(int i=0;i<block_num;i++){
        float cur=0;
        for(int j=0;j<THREAD_PER_BLOCK;j++){
            cur+=a[i*THREAD_PER_BLOCK+j];
        }
        res[i]=cur;
    }

    hipMemcpy(d_a,a,N*sizeof(float),hipMemcpyHostToDevice);

    dim3 Grid( N/THREAD_PER_BLOCK,1);
    dim3 Block( THREAD_PER_BLOCK,1);

    reduce1<float><<<Grid,Block>>>(d_a,d_out);

    hipMemcpy(out,d_out,block_num*sizeof(float),hipMemcpyDeviceToHost);

    if(check(out,res,block_num))printf("the ans is right\n");
    else{
        printf("the ans is wrong\n");
        for(int i=0;i<block_num;i++){
            printf("%lf ",out[i]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_out);
}
